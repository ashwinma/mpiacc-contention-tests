#include <stdio.h>
#include "cudacommon.h"
#include "OptionParser.h"
#include "ResultDatabase.h"

#include "misc_defs.h"

extern unsigned long long mpido;
extern unsigned long long cudado;
extern unsigned long long mpidone;
extern unsigned long long cudadone;
extern pthread_barrier_t mpitest_barrier;

// ****************************************************************************
// Function: addBenchmarkSpecOptions
//
// Purpose:
//   Add benchmark specific command line argument parsing.
//
//   -nopinned
//   This option controls whether page-locked or "pinned" memory is used.
//   The use of pinned memory typically results in higher bandwidth for data
//   transfer between host and device.
//
// Arguments:
//   op: the options parser / parameter database
//
// Returns:  nothing
//
// Programmer: Jeremy Meredith
// Creation: September 08, 2009
//
// Modifications:
//
// ****************************************************************************
void addBenchmarkSpecOptions(OptionParser &op)
{
    op.addOption("nopinned", OPT_BOOL, "",
                 "disable usage of pinned (pagelocked) memory", 'p');
}

// ****************************************************************************
// Function: runBenchmark
//
// Purpose:
//   Measures the bandwidth of the bus connecting the host processor to the
//   OpenCL device.  This benchmark repeatedly transfers data chunks of various
//   sizes across the bus to the host from the device and calculates the
//   bandwidth for each chunk size.
//
// Arguments:
//  resultDB: the benchmark stores its results in this ResultDatabase
//  op: the options parser / parameter database
//
// Returns:  nothing
//
// Programmer: Jeremy Meredith
// Creation: September 08, 2009
//
// Modifications:
//    Jeremy Meredith, Wed Dec  1 17:05:27 EST 2010
//    Added calculation of latency estimate.
//
// ****************************************************************************
void RunBenchmark(ResultDatabase &resultDB,
                  OptionParser &op)
{
    bool verbose = op.getOptionBool("verbose");
    bool pinned  = !op.getOptionBool("nopinned");
	int cur_device;
	hipGetDevice(&cur_device);
	CHECK_CUDA_ERROR();
	hipSetDevice(1);
	CHECK_CUDA_ERROR();

    // Sizes are in kb
    //int nSizes  = 20;
    //int sizes[20] = {1,2,4,8,16,32,64,128,256,512,1024,2048,4096,8192,16384,
	//	     32768,65536,131072,262144,524288};
    int nSizes  = 10;
    int sizes[10] = {1024,2048,4096,8192,16384,
		     32768,65536,131072,262144,524288};
    long long numMaxFloats = 1024 * (sizes[nSizes-1]) / 4;

    // Create some host memory pattern
    float *hostMem1;
    float *hostMem2;
    if (pinned)
    {
	    if (verbose) cout << "using pinned memory\n";
        hipHostMalloc((void**)&hostMem1, sizeof(float)*numMaxFloats);
        hipError_t err1 = hipGetLastError();
        hipHostMalloc((void**)&hostMem2, sizeof(float)*numMaxFloats);
        hipError_t err2 = hipGetLastError();
	while (err1 != hipSuccess || err2 != hipSuccess)
	{
	    // free the first buffer if only the second failed
	    if (err1 == hipSuccess)
	        hipHostFree((void*)hostMem1);

	    // drop the size and try again
	    if (verbose) cout << " - dropping size allocating pinned mem\n";
	    --nSizes;
	    if (nSizes < 1)
	    {
		cerr << "Error: Couldn't allocated any pinned buffer\n";
		return;
	    }
	    numMaxFloats = 1024 * (sizes[nSizes-1]) / 4;
            hipHostMalloc((void**)&hostMem1, sizeof(float)*numMaxFloats);
            err1 = hipGetLastError();
            hipHostMalloc((void**)&hostMem2, sizeof(float)*numMaxFloats);
            err2 = hipGetLastError();
	}
   }
    else
    {
        hostMem1 = new float[numMaxFloats];
        hostMem2 = new float[numMaxFloats];
    }
    for (int i=0; i<numMaxFloats; i++)
        hostMem1[i] = i % 77;

    float *device;
    hipMalloc((void**)&device, sizeof(float) * numMaxFloats);
    while (hipGetLastError() != hipSuccess)
    {
	// drop the size and try again
	if (verbose) cout << " - dropping size allocating device mem\n";
	--nSizes;
	if (nSizes < 1)
	{
	    cerr << "Error: Couldn't allocated any device buffer\n";
	    return;
	}
	numMaxFloats = 1024 * (sizes[nSizes-1]) / 4;
        hipMalloc((void**)&device, sizeof(float) * numMaxFloats);
    }

    hipMemcpy(device, hostMem1,
               numMaxFloats*sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    const unsigned int passes = op.getOptionInt("passes");
	hipStream_t d_stream;
	hipStreamCreate(&d_stream);
	CHECK_CUDA_ERROR();


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    CHECK_CUDA_ERROR();

	if(cudado != 1)
	{
	    int rc = pthread_barrier_wait(&mpitest_barrier);
		if(rc != 0 && rc != PTHREAD_BARRIER_SERIAL_THREAD)
		{
			printf("Could not wait on barrier\n");
			exit(-1);
	    }
	}
	
	int cur_dev;
	hipGetDevice(&cur_dev);
	CHECK_CUDA_ERROR();
    cout << "[CUDA-Task] Running CUDA benchmarks on device: " << cur_dev << "\n";
    // Three passes, forward and backward both
	int iter = 0;
#ifndef LONG_GPU_RUNS
	do {
#else
    for (int pass = 0; pass < passes; pass++)
    {
#endif
        cout << "Running benchmarks, pass: " << iter << "\n";
        // store the times temporarily to estimate latency
        //float times[nSizes];
        // Step through sizes forward on even passes and backward on odd
        for (int i = 0; i < nSizes; i++)
        {
            int sizeIndex;
            if ((iter % 2) == 0)
                sizeIndex = i;
            else
                sizeIndex = (nSizes - 1) - i;

            int nbytes = sizes[sizeIndex] * 1024;

            hipEventRecord(start, d_stream);
			if(pinned)
            	hipMemcpyAsync(hostMem2, device,
               			nbytes, hipMemcpyDeviceToHost, d_stream);
			else
            	hipMemcpy(hostMem2, device,
                       nbytes, hipMemcpyDeviceToHost);
            hipEventRecord(stop, d_stream);
            hipEventSynchronize(stop);
            float t = 0;
            hipEventElapsedTime(&t, start, stop);
            //times[sizeIndex] = t;

            // Convert to GB/sec
            if (verbose)
            {
                cerr << "size " <<sizes[sizeIndex] << "k took " << t <<
                        " ms\n";
            }

            double speed = (double(sizes[sizeIndex]) * 1024. / (1000*1000)) / t;
            char sizeStr[256];
            sprintf(sizeStr, "% 7dkB", sizes[sizeIndex]);
            resultDB.AddResult("ReadbackSpeed", sizeStr, "GB/sec", speed);
            resultDB.AddResult("ReadbackTime", sizeStr, "ms", t);
        }
		iter++;
	//resultDB.AddResult("ReadbackLatencyEstimate", "1-2kb", "ms", times[0]-(times[1]-times[0])/1.);
	//resultDB.AddResult("ReadbackLatencyEstimate", "1-4kb", "ms", times[0]-(times[2]-times[0])/3.);
	//resultDB.AddResult("ReadbackLatencyEstimate", "2-4kb", "ms", times[1]-(times[2]-times[1])/1.);
#ifdef LONG_GPU_RUNS
    }
	cudadone = 1;
#else
	} while(mpidone != 1);
#endif

    // Cleanup
	printf("Done with CUDA Tests...\n");
	hipStreamDestroy(d_stream);
	CHECK_CUDA_ERROR();
    hipFree((void*)device);
    CHECK_CUDA_ERROR();
    if (pinned)
    {
        hipHostFree((void*)hostMem1);
        CHECK_CUDA_ERROR();
        hipHostFree((void*)hostMem2);
        CHECK_CUDA_ERROR();
    }
    else
    {
        delete[] hostMem1;
        delete[] hostMem2;
        hipEventDestroy(start);
	    hipEventDestroy(stop);
    }
	hipSetDevice(cur_device);
    CHECK_CUDA_ERROR();
}
